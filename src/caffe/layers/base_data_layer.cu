#include <vector>

#include "caffe/data_layers.hpp"

namespace caffe {

template <typename Dtype>
void BasePrefetchingDataLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  // Batch<Dtype>* batch = prefetch_full_.pop("Data layer prefetch queue empty");
  // Reshape to loaded data.
  // LOG(INFO)<<"jq loadbatch";
  Batch<Dtype>* batch;

  batch= new Batch<Dtype>;
  batch->data_.ReshapeLike(prefetch_[0].data_);
  batch->label_.ReshapeLike(prefetch_[0].label_);
  load_batch(batch);
  // LOG(INFO)<<"jq loadbatch"<<batch->data_.shape_string();
  top[0]->ReshapeLike(batch->data_);
  // Copy the data
  caffe_copy(batch->data_.count(), batch->data_.gpu_data(),
      top[0]->mutable_gpu_data());
  // LOG(INFO)<<"jq loadbatch";
  if (this->output_labels_) {
    // Reshape to loaded labels.
    top[1]->ReshapeLike(batch->label_);
    // Copy the labels.
    caffe_copy(batch->label_.count(), batch->label_.gpu_data(),
        top[1]->mutable_gpu_data());
  }
  // LOG(INFO)<<"jq loadbatch";
  // Ensure the copy is synchronous wrt the host, so that the next batch isn't
  // copied in meanwhile.
  CUDA_CHECK(hipStreamSynchronize(hipStreamDefault));
  // prefetch_free_.push(batch);
  delete batch;
}

INSTANTIATE_LAYER_GPU_FORWARD(BasePrefetchingDataLayer);

}  // namespace caffe
